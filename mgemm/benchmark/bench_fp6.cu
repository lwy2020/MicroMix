#include "fp6.h"
#include "gemm.h"

#include "cutlass/numeric_conversion.h"

using namespace cute;

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::mx_float6_t<cutlass::float_e3m2_t>;    // Element type for A matrix operand

// B matrix configuration
using         ElementB    = cutlass::mx_float6_t<cutlass::float_e3m2_t>;    // Element type for B matrix operand

// C/D matrix configuration
using         ElementD    = cutlass::bfloat16_t;                            // Element type for D matrix operand
using         ElementC    = cutlass::bfloat16_t;                            // Element type for C matrix operand

int main() {
    const int M = 2048;
    const int N = 4096;
    const int K = 4096;
    const int block_size = 32;
    
    ElementA::DataType *A;
    ElementB::DataType *B;
    ElementC *C;
    ElementD *D;
    A = new ElementA::DataType[M * K];
    B = new ElementB::DataType[N * K];
    C = new ElementC[M * N];
    D = new ElementD[M * N];
    
    // 创建 scale 数组（每 block_size 个元素对应一个缩放因子）
    int szA = ((M * K + block_size - 1) / block_size);
    ElementA::ScaleFactorType *scaleA = new ElementA::ScaleFactorType[((M * K + block_size - 1) / block_size)];
    int szB = ((N * K + block_size - 1) / block_size);
    ElementB::ScaleFactorType *scaleB = new ElementB::ScaleFactorType[((N * K + block_size - 1) / block_size)];
    

    std::srand(static_cast<unsigned int>(std::time(0)));
    cutlass::NumericConverter<ElementA::DataType, float, cutlass::FloatRoundStyle::round_to_nearest> converterA;
    cutlass::NumericConverter<ElementB::DataType, float, cutlass::FloatRoundStyle::round_to_nearest> converterB;
    cutlass::NumericConverter<ElementA::ScaleFactorType, float, cutlass::FloatRoundStyle::round_to_nearest> converterSFA;
    cutlass::NumericConverter<ElementB::ScaleFactorType, float, cutlass::FloatRoundStyle::round_to_nearest> converterSFB;
    
    for (int i = 0; i < M * K; ++i) {
        // 模拟浮点值
        float f = static_cast<float>(std::rand()) / RAND_MAX * 56.0f - 28.0f;
        
        // 这里可以使用 CUTLASS 的量化转换器（如果你使用完整的库）
        // 否则使用构造函数转换
        A[i] = converterA(f);
    }

    for (int i = 0; i < M * N; ++i) {
        // 模拟浮点值
        ElementC f = static_cast<ElementC>(12.0 * std::rand() / RAND_MAX - 6.0);
        
        // 这里可以使用 CUTLASS 的量化转换器（如果你使用完整的库）
        // 否则使用构造函数转换
        C[i] = f;
    }
    for (int i = 0; i < N * K; ++i) {
        // 模拟浮点值
        float f = static_cast<float>(std::rand()) / RAND_MAX * 56.0f - 28.0f;
        
        // 这里可以使用 CUTLASS 的量化转换器（如果你使用完整的库）
        // 否则使用构造函数转换
        B[i] = converterB(f);
    }


    // 随机初始化 scale（每 block 一个）
    for (size_t i = 0; i < szA; ++i) {
        scaleA[i] = converterSFA(static_cast<float>(std::rand()) / RAND_MAX * 255.0f);  // [0.1, 1.0]
    }
    for (size_t i = 0; i < szB; ++i) {
        scaleB[i] = converterSFB(static_cast<float>(std::rand()) / RAND_MAX * 255.0f);  // [0.1, 1.0]
    }
    ElementA::DataType *A_d;
    ElementB::DataType *B_d;
    ElementC *C_d;
    ElementD *D_d;    
    ElementA::ScaleFactorType *SFA_d;
    ElementB::ScaleFactorType *SFB_d;

    hipMalloc((void**)&A_d, M * K * sizeof(ElementA::DataType));
    hipMalloc((void**)&B_d, N * K * sizeof(ElementB::DataType));
    hipMalloc((void**)&C_d, M * N * sizeof(ElementC));
    hipMalloc((void**)&D_d, M * N * sizeof(ElementD));
    hipMalloc((void**)&SFA_d, szA * sizeof(ElementA::ScaleFactorType));
    hipMalloc((void**)&SFB_d, szB * sizeof(ElementB::ScaleFactorType));
    hipMemcpy(A_d, A, M * K * sizeof(ElementA::DataType), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N * K * sizeof(ElementB::DataType), hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, M * N * sizeof(ElementC), hipMemcpyHostToDevice);
    hipMemcpy(SFA_d, scaleA, szA * sizeof(ElementA::ScaleFactorType), hipMemcpyHostToDevice);
    hipMemcpy(SFB_d, scaleB, szB * sizeof(ElementB::ScaleFactorType), hipMemcpyHostToDevice);
    
    
    // Timing using CUDA events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    for (int it = 0; it < 200; it ++) {
        matmul_host6(A_d, B_d, M, N, K, C_d, D_d, SFA_d, SFB_d);
    }
    
    // timed runs
    const int timed_iters = 400;
    CHECK_CUDA(hipEventRecord(start));
    for (int it = 0; it < timed_iters; it ++) {
        matmul_host6(A_d, B_d, M, N, K, C_d, D_d, SFA_d, SFB_d);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    hipMemcpy(D, D_d, M * N * sizeof(ElementD), hipMemcpyDeviceToHost);

    double time_sec = static_cast<double>(milliseconds) / 1000.0;
    
    double flops_per_gemm = 2.0 * static_cast<double>(M) * static_cast<double>(N) * static_cast<double>(K);
    double total_flops = flops_per_gemm * static_cast<double>(timed_iters);
    double gflops = total_flops / (time_sec * 1.0e9);
    double tflops = total_flops / (time_sec * 1.0e12);

    std::printf("GEMM average time per run: %.3f ms\n", milliseconds / timed_iters);
    std::printf("Total runs: %d, problem size M=%d N=%d K=%d\n", timed_iters, M, N, K);
    std::printf("FLOPs per run: %.0f\n", flops_per_gemm);
    std::printf("Total FLOPs (all runs): %.0f\n", total_flops);
    std::printf("Achieved performance: %.3f TFLOPS\n", tflops);
    std::cout << "mxfp6 gemm finished." << std::endl;
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipFree(D_d);
    hipFree(SFA_d);
    hipFree(SFB_d);
    return 0;
}